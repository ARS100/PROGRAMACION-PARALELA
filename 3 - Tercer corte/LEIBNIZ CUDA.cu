// 2182059 - Alejandro Romero Serrano

//Librerias utilizadas

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N 1000000000

//función global de leibniz
__global__ void pi_elem(float *values)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    values[i] = pow((float)-1,(float)i);
    values[i] /= (2*i+1);
}

int main(int argc, char *argv[]) {
    float pi = 0;
    int i;
    float *d_values, *h_values;

    hipMalloc((void**)&d_values,N*sizeof(float));
    h_values = (float*)malloc(N*sizeof(float));

    pi_elem <<< N/32, 32 >>>(d_values);

    hipMemcpy(h_values,d_values,N*sizeof(float), hipMemcpyDeviceToHost);

    printf("%f\n", h_values[0]);
    for (i = 0; i < N; i++) {
        pi += h_values[i];
    }
    printf("%f\n", 4*pi);
    return 0;
}
