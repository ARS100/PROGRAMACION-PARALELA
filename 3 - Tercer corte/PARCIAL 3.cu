
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <fstream>


#include "math.h"

#define N 3   // Se definen 3 hilos de trabajo únicamente, aunque
              // también se puede considerar al numero de semillas como
              // la cantidad de hilos. 

using namespace std;
using std::ofstream;
using std::endl; using std::string;
using std::fstream;

// Acá quedan las estructuras y generación de semillas
// apoyadas mediante el código del profesor.

typedef struct{
    float x; 
    float y;
    float t; 
}posParticle;



__device__ void generarSemillas(posParticle *semillas, const  int maxParticles) 
{

    unsigned int num_semillas = maxParticles;
    for (int i = 0; i < num_semillas; ++i)
    {
        const float radio = 0.1;
        const float alfa  = 2.0f * M_PI * (float)i / (float)num_semillas;
        semillas[i].t = 0.0;
        semillas[i].x = 0.5f + radio*cos(alfa);
        semillas[i].y = 0.5f + radio*sin(alfa);
    }
    
}

__global__ void iniciarYterminarIntegral(double t, posParticle *se, float *dev_x, float *dev_y, float *dev_t){
    generarSemillas(se,N);

    // se asignan el valor inicial y final del tiempo a
    // el puntero referenciado de arriba semilla, conectado a
    // esta función como 'se'.

    se[0].t = 0;
    se[N].t = 1;

    // Se declararán nuevos punteros para asignar v1
    // correspondientes a la posición posterior a la integral
    // ejecutada sobre cada valor.

    double *X = new double[(int)N];
    double *Y = new double[(int)N];
    
    // Este es el ID que será recorrido
    // para cada uno de los valores de asignación
    // para los vectores posición inicial.
    int h;

    
    double *coordenadasX = new double[(int)N];
    double *coordenadasY = new double[(int)N];

    dev_x[0] = se[0].x;
    dev_y[0] = se[0].y;
    dev_t[0] = se[0].t;
    

    int columna = threadIdx.x; 
    //int fila = threadIdx.y; 

    //int pos = columna + N*fila; 

    
    h = (t)/(N);
    se[columna].t = t;
    if(columna < N){
      double v1 = sin(se[columna].t*M_PI/180);
        // se[columna].x = v1;
      X[columna] = v1;
      double v2 = cos(se[columna].t*M_PI/180);
        // se[columna].y = v2;
      Y[columna] = v2;
    }
    
    // inmediatamente después a cada valor comprendido
    // entre el 0 y 1, le asignamos su integral y se 
    // identificará como la nueva coordenada.


    double booleX = ((7 * X[columna] + 32 * X[columna+h] +
                12 * X[columna+2*h] +32 * X[columna+3*h]
                + 7 * X[columna+4*h] *
                 2 * h / 45));
    dev_x[columna] = dev_x[columna] + booleX;

    // La regla de integración aplicada es la regla de Boole.

    double booleY = ((7 * Y[columna] + 32 * Y[columna+h] +
                12 * Y[columna+2*h] +32 * Y[columna+3*h]
                + 7 * Y[columna+4*h] *
                 2 * h / 45));
    dev_y[columna]= dev_y[columna] + booleY;

    
   
}


// Sigue el main, donde visualizaremos cada función paralelizada
// desde las declaraciones y el direccionamiento entre host y device
// como la llamada al kernel y las asignaciones correspondientes para
// posteriormente imprimir.

int main(){
    
    // Se DECLARAN los espacios en memoria CPU y GPU.

    int tiempo;
    cout << "Ingrese el tiempo correspondiente a utilizarse (es decir, la cantidad de intervalos): ";
    cin >> tiempo;
    float *hst_x, *hst_y, *hst_t; 
    float *dev_x, *dev_y, *dev_t; 

    // Se hace la RESERVA en el host.
    hst_x = (float*)malloc(N*sizeof(float));
    hst_y = (float*)malloc(N*sizeof(float));
    hst_t = (float*)malloc(N*sizeof(float));
    
    // Se hace la RESERVA en el device.
    hipMalloc( (void**)&dev_x, N*sizeof(float));
    hipMalloc( (void**)&dev_y, N*sizeof(float));
    hipMalloc( (void**)&dev_t, N*sizeof(float));

    // Se inicializan los espacios asignados anteriormente.
    for(int i=0; i<tiempo; i++) 
    { 
        dev_t[i] =  (float)i; 
    }
    // Copiamos los datos y los obtenemos tanto en CPU como en GPU.
    hipMemcpyToSymbol(HIP_SYMBOL(dev_x), hst_x, N*sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_y), hst_y, N*sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_t), hst_t, N*sizeof(float));

    // lanzamiento de kernel
    iniciarYterminarIntegral<<<1,N>>>(1, 0, dev_x, dev_y, dev_t); 
    for(int i=0; i<tiempo; i++) 
    { 
        hst_t[i] =  (float)i; 
    }
    // copiar los datos recogidos.
    hipMemcpy( hst_x, dev_x, N*sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( hst_y, dev_y, N*sizeof(float), hipMemcpyDeviceToHost );
    
    printf("Resultados:\n");
    printf("Intervalos de tiempo: ");
    cout << tiempo;

    string archivo = "prueba.txt";
    ofstream salida(archivo);

    //salida.open("/content/drive/MyDrive/prueba.txt");
    
    for(int i=0; i<tiempo; i++) 
    { 
      for(int j=0; j<N;j++) 
      {
           if(i==0){
           }
        
       cout << " \n En el tiempo # " << i << " se tiene en x, y, t respectivamente: "; 
       cout << hst_x[j] << "   ";
       cout << hst_y[j] << "   ";
       cout << hst_t[j] << "   ";
       cout << "\n";

        salida << " \n En el tiempo #  se tiene en x, y, t respectivamente: ";
        salida << hst_x[j] << "   ";
        salida << hst_y[j] << "   ";
        salida << hst_t[j] << "   ";
        salida << "\n";     
     
      } 
    } 
    salida.close();
    system("pause");
    return 0;
}
