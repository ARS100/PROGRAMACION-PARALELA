#include "hip/hip_runtime.h"
// 2182059 - Alejandro Romero Serrano

#include <iostream>
#include <vector>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

using namespace std;

#define N 1


//Función global de la generación de la matriz de la EC de calor para ser llamada en el MAIN como Kernel.

__global__ void sacarMatrizEcDeCalor(int **M, float k, float sT, float sX, float Ui, float Uf,float Uc, int Filas, int Columnas, int n)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    M = (int**)malloc(Filas*sizeof(int*));
    M[j] = (int*)malloc(Columnas*sizeof(int)); 
    
    float lamda = k*sT /pow(sX,2);
    // Acá se deben obtener los valores correspondientes:
    M[i][j] = Ui;
      if(j==0)
        {
         M[i][0]=Uc;
        }
        if(j==Columnas-1){
        M[i][Columnas-1]=Uf;
        }
        if(j!=0 & i!=0 & j!=Columnas-1)
        {
          M[i][j] = M[i-1][j] + lamda*(M[i-1][j+1]-2*M[i-1][j]+M[i-1][j-1]);
        }
        if(M[i][j]< 0.15)
        {
          M[i][j] = 0.1;
               if(M[i][j]< 0.1){
                   M[i][j] = 0.05;
               }
               if(M[i][j]< 0.01){
                   M[i][j] = 0.005;
               }
        }
    
}

int main(int argc, char *argv[]) {
    

    // Se inicializan los punteros para asignar valores en D,
    // con un tamaño de N. Estos se inicializan al igual que los de H.

    int **Valores_Matriz, **Valores_H;

    
    int Filas = 15;
    int Columnas = 20;
    
    hipMalloc((void**)&Valores_Matriz,Filas*sizeof(int));
    Valores_H = (int**)malloc(Columnas*sizeof(int));


    // Se llama hace el llamado al Kernel, con un hilo por cada
    // bloque de ejecución correspondiente a la cantidad de iteraciones.

    sacarMatrizEcDeCalor <<< N, 2 >>>(Valores_Matriz, 10, 1, 2, 15, 20, 17, Filas, Columnas, 2);

    // Se copia en el host los valores generados desde GPU.
    hipMemcpy(Valores_H,Valores_Matriz,N*sizeof(int),hipMemcpyDeviceToHost);

    // Este es el primer valor tras el comienzo de las iteraciones
    for(int i = 0; i< Filas; i++)
    {
        for(int j = 0; j< Columnas; j++){
          printf("%f\t", Valores_H[i][j]);
        }
        printf("%f\n");
    }
  




    return 1;
}
